#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "gtest/gtest.h"

__global__ void exec_kernel()
{
    __shared__ char char_ptr[16];

    extern __shared__ char char_ptr_dyn[];

    printf("\n char_ptr= %p, char_ptr+16= %p, char_ptr_dyn= %p\n",
           (void*)char_ptr,
           (void*)(char_ptr + 16),
           (void*)char_ptr_dyn);
}

TEST(Test, exe)
{
    exec_kernel<<<1, 1>>>();
    auto err = hipDeviceSynchronize();
    EXPECT_EQ(err, hipSuccess);
}


int main(int argc, char** argv)
{
    ::testing::InitGoogleTest(&argc, argv);

    return RUN_ALL_TESTS();
}
